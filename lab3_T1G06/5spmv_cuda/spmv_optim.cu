#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define N 1024
#define ROWSIZE 9
#define THREADS_PER_BLOCK 128

__global__ void cuspmv(int m, double* dvals, int* dcols, double* dx, double* dy) {
    __shared__ double s_vals[THREADS_PER_BLOCK * ROWSIZE];
    __shared__ int s_cols[THREADS_PER_BLOCK * ROWSIZE];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    if (i < m) {
        double partial_sum = 0.0;

        // Load dvals and dcols into shared memory
        for (int j = 0; j < ROWSIZE; j++) {
            int base_idx = i * ROWSIZE + j;
            s_vals[tid + j * blockDim.x] = dvals[base_idx];
            s_cols[tid + j * blockDim.x] = dcols[base_idx];
            __syncthreads();
            int col_idx = s_cols[tid + j * blockDim.x];
            partial_sum += s_vals[tid + j * blockDim.x] * dx[col_idx];
        }

        dy[i] = partial_sum;
    }
}

void spmv_cpu(int m, int r, double* vals, int* cols, double* x, double* y) {
    for(int i = 0; i < m; i++) {
        double partial_sum = 0.0;
        for(int j = 0; j < r; j++) {
            int idx = i*r + j;
            partial_sum += vals[idx] * x[cols[idx]];
        }
        y[i] = partial_sum;
    }
}

void fill_matrix(double* vals, int* cols) {
    int indx[ROWSIZE];
    int row_count = 0;
    for(int j = 0; j < N ; j++) {
        for(int i = 0; i < N; i++) {
            indx[0] = i     + (j - 2)*N;
            indx[1] = i     + (j - 1)*N;
            indx[2] = i - 2 +  j     *N;
            indx[3] = i - 1 +  j     *N;
            indx[4] = i     +  j     *N;
            indx[5] = i + 1 +  j     *N;
            indx[6] = i + 2 +  j     *N;
            indx[7] = i     + (j + 1)*N;
            indx[8] = i     + (j + 2)*N;

            for(int row = 0; row < ROWSIZE; row++) {
                if(indx[row] < 0 || indx[row] >= N*N) {
                    cols[row + row_count*ROWSIZE] = i + j*N;
                    vals[row + row_count*ROWSIZE] = 0.0;
                } else {
                    cols[row + row_count*ROWSIZE] = indx[row];
                    if(row == 4) {
                        vals[row + row_count*ROWSIZE] = 0.95;
                    } else {
                        vals[row + row_count*ROWSIZE] = -0.95/(ROWSIZE - 1);
                    }
                }
            }
            row_count++;
        }
    }
    vals[4 + (N*N/2 + N/2)*ROWSIZE] =  1.001*vals[4 + (N*N/2 + N/2)*ROWSIZE];
}

int main() {
    int vec_size = N*N;

    float time_cpu, time_gpu;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double* x     = (double*) malloc (vec_size*sizeof(double));
    double* y_cpu = (double*) malloc (vec_size*sizeof(double));
    double* y_gpu = (double*) malloc (vec_size*sizeof(double));

    double* Avals = (double*) malloc (ROWSIZE*vec_size*sizeof(double));
    int*    Acols = (int   *) malloc (ROWSIZE*vec_size*sizeof(int));

    // fill vector with sinusoidal for testing the code
    for(int i = 0; i < vec_size; i++) {
        x[i] = sin(i*0.01);
        y_cpu[i] = 0.0;
    }

    fill_matrix(Avals, Acols);

    // measure time of CPU implementation
    hipEventRecord(start);
    for (int i = 0; i < 100; ++i)
        spmv_cpu(vec_size, ROWSIZE, Avals, Acols, x, y_cpu);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_cpu, start, stop);

    // allocate arrays in GPU
    double *dx, *dy_gpu, *dAvals;
    int *dAcols;

    hipMalloc((void**)&dx, vec_size * sizeof(double));
    hipMalloc((void**)&dy_gpu, vec_size * sizeof(double));
    hipMalloc((void**)&dAvals, ROWSIZE * vec_size * sizeof(double));
    hipMalloc((void**)&dAcols, ROWSIZE * vec_size * sizeof(int));

    // transfer data to GPU
    hipMemcpy(dx, x, vec_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dAvals, Avals, ROWSIZE * vec_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dAcols, Acols, ROWSIZE * vec_size * sizeof(int), hipMemcpyHostToDevice);

    // calculate threads and blocks
    int blocksPerGrid = (vec_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // measure time of GPU implementation
    hipEventRecord(start);
    for (int i = 0; i < 100; ++i)
        cuspmv<<<blocksPerGrid, THREADS_PER_BLOCK>>>(vec_size, dAvals, dAcols, dx, dy_gpu);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_gpu, start, stop);

    // transfer result to CPU RAM
    hipMemcpy(y_gpu, dy_gpu, vec_size * sizeof(double), hipMemcpyDeviceToHost);

    // free arrays in GPU
    hipFree(dx);
    hipFree(dy_gpu);
    hipFree(dAvals);
    hipFree(dAcols);

    // comparison between gpu and cpu results
    double norm2 = 0.0;
    for(int i = 0; i < vec_size; i++)
        norm2 += (y_cpu[i] - y_gpu[i])*(y_cpu[i] - y_gpu[i]);

    norm2 = sqrt(norm2);

    printf("spmv comparison cpu vs gpu error: %e, size %d\n", norm2, vec_size);
    printf("CPU Time: %lf\n", time_cpu/1000);
    printf("GPU Time: %lf\n", time_gpu/1000);

    // free CPU arrays
    free(x);
    free(y_cpu);
    free(y_gpu);
    free(Acols);
    free(Avals);

    return 0;
}
